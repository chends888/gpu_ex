#include "hip/hip_runtime.h"
// how to compile: nvcc -std=c++11 FILE -o EXECUTABLE


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <math.h>

int main () {
    thrust::host_vector<double> host(5);
    thrust::host_vector<double> mean(5);
    thrust::fill(thrust::host, mean.begin(), mean.end(), 500);
    std::cout << "ok";
    // fill host vector with numbers
    for (int i=0; i<host.size(); i++) {
        host.push_back(i);
    }
    // calculate variance
    auto start = std::chrono::high_resolution_clock::now();
    for (int i=0; i<host.size(); i++) {
        host[i] = host[i] - mean[i];
    }
    double sum = 0;
    for (int i=0; i<host.size(); i++) {
        sum += pow(host[i], 2);
    }
    auto finish = std::chrono::high_resolution_clock::now();
    auto time_span = (std::chrono::duration_cast<std::chrono::duration<double>>(finish-start)).count();
    std::cout << "Time using 2 vecs: " << time_span << " s" << std::endl;
    std::cout << "Variance: " << sum << std::endl;

    start = std::chrono::high_resolution_clock::now();
    sum = 0;
    thrust::transform(host.begin(), host.end(),
                      thrust::make_constant_iterator(500),
                      host.begin(),
                      thrust::minus<double>());
    for (int i=0; i<host.size(); i++) {
        sum += pow(host[i], 2);
    }
    finish = std::chrono::high_resolution_clock::now();
    time_span = (std::chrono::duration_cast<std::chrono::duration<double>>(finish-start)).count();
    std::cout << "Time using constant iterator: " << time_span << " s" << std::endl;
    std::cout << "Variance: " << sum << std::endl;

}
