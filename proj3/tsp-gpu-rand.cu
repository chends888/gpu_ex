#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
#include <math.h>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#define ITER 10000



__global__ void calc_dists(double *xpos, double *ypos, double *dists, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    dists[i * N + j] = sqrt(pow((xpos[i] - xpos[j]), 2) + pow((ypos[i] - ypos[j]), 2));
}

__global__ void calc_path_dists(int *all_paths, double* path_dists, double *dists, int N) {
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_i >= N * ITER) return;

    hiprandState st;
    hiprand_init(0, t_i, 0, &st);
    for (int i=0; i<N; i++) {
        all_paths[(t_i * N) + i] = i;
    }

    for (int i=1; i<N; i++) {
        int tmp = all_paths[N * t_i + i];
        int rand_i = (int) ((N - i) * hiprand_uniform(&st) + i);
        all_paths[N * t_i + i] = all_paths[N * t_i + rand_i];
        all_paths[N * t_i + rand_i] = tmp;
    }

    double path_dist;
    for (int i=0; i<N; i++) {
        path_dist += dists[all_paths[t_i * N + i] * N + all_paths[t_i * N + i + 1]];
    }
    path_dist += dists[all_paths[N - 1]];
    path_dists[t_i] = path_dist;
    //path_dists[t_i] += dists[all_paths[N - 1]];

}

int main() {
    int N;
    std::cin >> N;
    //long steps = 5000;

    thrust::host_vector<double> xpos(N), ypos(N);
    //thrust::host_vector<int> all_paths(N * steps);
    double x, y;

    for (int i=0; i<N; i++) {
        std::cin >> x;
        std::cin >> y;
        xpos[i] = x;
        ypos[i] = y;
        //points[i] = i;
    }
    //for (int i=0; i<N; i++) {
    //    std::cout << xpos[i] << " ";
    //    std::cout << ypos[i] << " ";
    //}
    //std::cout << std::endl;
    //for (int i=0; i<N*N; i++) {
    //    std::cout << dists[i] << " ";
    //}
    //std::cout << std::endl;
    //hipMalloc((void **) &xpos, sizeof(double) * N);
    //hipMalloc((void **) &ypos, sizeof(double) * N);
    //hipMalloc((void **) &dists, sizeof(double) * N * N);
    thrust::device_vector<double> xpos_d(xpos), ypos_d(ypos), dists_d(N * N), path_dists_d(ITER);
    thrust::device_vector<int> all_paths_d(N * ITER);

    dim3 threads(32, 32);
    dim3 grid(N / threads.x, N / threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);

    calc_dists<<<grid, threads>>>(thrust::raw_pointer_cast(xpos_d.data()),
                                  thrust::raw_pointer_cast(ypos_d.data()),
                                  thrust::raw_pointer_cast(dists_d.data()),
                                  N);
    calc_path_dists<<<ceil(ITER/1024.0), 1024>>>(thrust::raw_pointer_cast(all_paths_d.data()),
                                 thrust::raw_pointer_cast(path_dists_d.data()),
                                 thrust::raw_pointer_cast(dists_d.data()),
                                    N);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    thrust::device_vector<double>::iterator min = thrust::min_element(path_dists_d.begin(), path_dists_d.end());
    int min_idx = min - path_dists_d.begin();
    double min_val = *min;
    //printf("%d\n", min_idx);
    printf("%f 0\n", min_val);

    for (int i=min_idx*N; i<(min_idx+1)*N; i++) {
        std::cout << all_paths_d[i] << " ";
    }
    printf("\n");

    std::cerr << msecTotal/1000 << " s" << std::endl;

}
