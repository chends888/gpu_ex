#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
#include <math.h>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#define SIZE 1

__global__ void calc_dists(double *xpos, double *ypos, double *dists, int N) {
    //printf("%d", N);
    //int test = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("%d, %d \n", i, j);
    dists[i*N+j] = sqrt(pow((xpos[i] - xpos[j]), 2) + pow((ypos[i] - ypos[j]), 2));
    //printf("%f ", dists[i*N+j]);
}

__global__ void shuff_vecs(double *xpos, double *ypos, int *points, int N) {
    int t_i = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState st;
    hiprand_init(0, 1, 0, &st);
    //N = 4;

    for (int i=1; i<N; i++) {
        double tmp = points[i];
        //double tmp_y = points[i];
        //double new_x = xpos[(int) ((N-i) * hiprand_uniform(&st) + i)];
        int rand_i = (int) ((N-i) * hiprand_uniform(&st) + i);
        //posx[i] = posx[rand_i];
        points[i] = points[rand_i];
        points[rand_i] = tmp;
        //posy[rand_i] = tmp_y;
        //printf("%d, ", test);
    }
    for (int i=0; i<N; i++) {
        printf("%d, ", points[i]);
    }
}

int main() {
    int N;
    std::cin >> N;

    thrust::host_vector<double> xpos(N), ypos(N), dists(N * N), points(N);
    double x, y;

    for (int i=0; i<N; i++) {
        std::cin >> x;
        std::cin >> y;
        xpos[i] = x;
        ypos[i] = y;
        points[i] = i;
    }
    //for (int i=0; i<N; i++) {
    //    std::cout << xpos[i] << " ";
    //    std::cout << ypos[i] << " ";
    //}
    //std::cout << std::endl;
    //for (int i=0; i<N*N; i++) {
    //    std::cout << dists[i] << " ";
    //}
    //std::cout << std::endl;
    //hipMalloc((void **) &xpos, sizeof(double) * N);
    //hipMalloc((void **) &ypos, sizeof(double) * N);
    //hipMalloc((void **) &dists, sizeof(double) * N * N);
    thrust::device_vector<double> xpos_d(xpos), ypos_d(ypos), dists_d(dists);
    thrust::device_vector<int> points_d(points);

    dim3 threads(32, 32);
    dim3 grid(N / threads.x, N / threads.y);
    calc_dists<<<grid, threads>>>(thrust::raw_pointer_cast(xpos_d.data()),
                                  thrust::raw_pointer_cast(ypos_d.data()),
                                  thrust::raw_pointer_cast(dists_d.data()),
                                  N);
    shuff_vecs<<<SIZE, 1>>>(thrust::raw_pointer_cast(xpos_d.data()),
                               thrust::raw_pointer_cast(ypos_d.data()),
                               thrust::raw_pointer_cast(points_d.data()),
                               N);
    //dim3 threads(1024);
    //dim3 grid(SIZE);

}

