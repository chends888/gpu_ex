#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main() {
    int N;
    std::cin >> N;

    thrust::host_vector<double> xpos(N), ypos(N), dists(N * N);
    double x, y;

    for (int i=0; i<N; i++) {
        // std::cin >> x;
        // std::cin >> y;
        x = 1;
        y = 2;
        xpos.push_back(x);
        ypos.push_back(y);
    }
    for (int i=0; i<N; i++) {
        std::cout << xpos[i] << " ";
        std::cout << ypos[i] << " ";
    }
    std::cout << std::endl;
    //for (int i=0; i<N*N; i++) {
    //    std::cout << dists[i] << " ";
    //}
    //std::cout << std::endl;
    //hipMalloc((void **) &xpos, sizeof(double) * N);
    //hipMalloc((void **) &ypos, sizeof(double) * N);
    //hipMalloc((void **) &dists, sizeof(double) * N * N);

}
